#include "hip/hip_runtime.h"
// includes, system
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

// includes, project
#include <cutil.h>

// includes, kernel
#include <dA_kernel.cu>


#define N_FEATS 20
#define N_OBS 10


// declarations for CPU train functions
extern "C"
void dA_train_gold(dA*, int*, double, double);
void dA_get_hidden_values(dA*, int*, double*);
void dA_get_reconstructed_input(dA*, double*, double*);


// functions defined in this file are for intializing the autoencoder
double uniform(double min, double max);
void dA__construct(dA *model, int N, int n_visible, int n_hidden, double **W, double *hbias, double *vbais);
void dA__destruct(dA *model);
void dA_reconstruct(dA *model, int *x, double *z);
void test_dbn();
void dA_train_on_device(dA*, int[][N_FEATS], double, double);


// Begin definign functions
double uniform(double min, double max) {
  return rand() / (RAND_MAX + 1.0) * (max - min) + min;
}


void dA__construct(dA* model, int N, int n_visible, int n_hidden, double **W, double *hbias, double *vbias) {
  int i, j;
  double a = 1.0 / n_visible;

  model->N = N;
  model->n_visible = n_visible;
  model->n_hidden = n_hidden;

  if(W == NULL) {
    model->W = (double **)malloc(sizeof(double*) * n_hidden);
    model->W_flat = (double*)malloc(sizeof(double)*n_hidden*n_visible);
    model->W[0] = (double *)malloc(sizeof(double) * n_visible * n_hidden);
    for(i=0; i<n_hidden; i++) model->W[i] = model->W[0] + i * n_visible;

    for(i=0; i<n_hidden; i++) {
      for(j=0; j<n_visible; j++) {
        double u = uniform(-a, a);
        model->W_flat[i*n_visible + j] = u;
        model->W[i][j] = u;
      }
    }
  } else {
    model->W = W;
  }

  if(hbias == NULL) {
    model->hbias = (double *)malloc(sizeof(double) * n_hidden);
    for(i=0; i<n_hidden; i++) model->hbias[i] = 0;
  } else {
    model->hbias = hbias;
  }

  if(vbias == NULL) {
    model->vbias = (double *)malloc(sizeof(double) * n_visible);
    for(i=0; i<n_visible; i++) model->vbias[i] = 0;
  } else {
    model->vbias = vbias;
  }
}


void dA__destruct(dA* model) {
  free(model->W[0]);
  free(model->W);
  free(model->W_flat);
  free(model->hbias);
  free(model->vbias);
}


void dA_reconstruct(dA* model, int *x, double *z) {
  double *y = (double *)malloc(sizeof(double) * model->n_hidden);

  dA_get_hidden_values(model, x, y);
  dA_get_reconstructed_input(model, y, z);

  free(y);
}


int* flatten_array(int arr[N_OBS][N_FEATS]) {
  int *flat = (int *)malloc(sizeof(int) * N_OBS * N_FEATS);
  for (int i=0; i < N_OBS; ++i) {
    for (int j=0; j < N_FEATS; ++j) {
      flat[i*N_FEATS + j] = arr[i][j];
    }
  }
  return flat;
}

double* flatten_w(double **W, int n_visible, int n_hidden) {
  double *flat = (double *)malloc(sizeof(double) * n_visible * n_hidden);
  for (int i=0; i < n_hidden; ++i) {
    for (int j=0; j < n_visible; ++j) {
      flat[i*n_visible + j] = W[i][j];
    }
  }
  return flat;
}

int * allocate_device_x() {
  int *x_d = NULL;
  int size = N_OBS * N_FEATS * sizeof(int);
  hipMalloc((void**)&x_d, size);
  return x_d;
}

dA init_device_ae(const dA model_h) {
  // allocate space
  dA model_d;
  model_d.N = model_h.N;
  model_d.n_visible = model_h.n_visible;
  model_d.n_hidden = model_h.n_hidden;
  model_d.hbias = NULL;
  model_d.vbias = NULL;
  model_d.W = NULL;
  model_d.W_flat = NULL;

  int W_size = sizeof(double) * model_h.n_hidden * model_h.n_visible;
  int hbias_size = sizeof(double) * model_h.n_hidden;
  int vbias_size = sizeof(double) * model_h.n_visible;

  hipMalloc((void**)&model_d.W_flat, W_size);
  hipMalloc((void**)&model_d.hbias, hbias_size);
  hipMalloc((void**)&model_d.vbias, vbias_size);

  // flatten w
  //double *flat_w = flatten_w(model_h.W, model_h.n_visible, model_h.n_hidden);

  // copy over data
  hipMemcpy(model_d.W_flat, model_h.W_flat, W_size, hipMemcpyHostToDevice);
  hipMemcpy(model_d.hbias, model_h.hbias, hbias_size, hipMemcpyHostToDevice);
  hipMemcpy(model_d.vbias, model_h.vbias, vbias_size, hipMemcpyHostToDevice);
  
  //free(flat_w);
  return model_d;
}

void copy_x_to_device(int *x_d, int *x_h) {
  int size = N_OBS * N_FEATS * sizeof(int);
  hipMemcpy(x_d, x_h, size, hipMemcpyHostToDevice);
}


void copy_ae_from_device(dA *model_h, const dA model_d) {
  int W_size = sizeof(double) * model_h->n_hidden * model_h->n_visible;
  int hbias_size = sizeof(double) * model_h->n_hidden;
  int vbias_size = sizeof(double) * model_h->n_visible;

  hipMemcpy(model_h->W_flat, model_d.W_flat, W_size, hipMemcpyDeviceToHost);
  hipMemcpy(model_h->hbias, model_d.hbias, hbias_size, hipMemcpyDeviceToHost);
  hipMemcpy(model_h->vbias, model_d.vbias, vbias_size, hipMemcpyDeviceToHost);
}

void free_device(dA *model) {
  hipFree(model->W_flat);
  hipFree(model->hbias);
  hipFree(model->vbias);
  model->W_flat = NULL;
  model->hbias = NULL;
  model->vbias = NULL;
}
  

void dA_train_on_device(dA *model_h, int train_X[][N_FEATS], double learning_rate, double corruption_level) {
  // call kernel function from here
  // assign one observation to each block, each thread parallelizes a feature
  
  // flatten input array
  int *X_h = flatten_array(train_X);

  // allocate space on device
  int *X_d = allocate_device_x();
  dA model_d = init_device_ae(*model_h);

  // copy data over to device
  copy_x_to_device(X_d, X_h);
  //  copy_ae_to_device(model_d, model_h);

  // define kernel dimensions
  int batch_size = 1;
  dim3 dim_grid(batch_size, 1, 1);
  dim3 dim_block(N_FEATS, 1, 1);
  dA_train_kernel<<<dim_grid, dim_block>>>(model_d, X_d, learning_rate, corruption_level);
  hipDeviceSynchronize();
  
  // read results from device
  copy_ae_from_device(model_h, model_d);

  // free up memory
  free(X_h);
  free_device(&model_d);
}



void test_dbn(void) {
  srand(0);
  int i, j, epoch;

  double learning_rate = 0.1;
  double corruption_level = 0.3;
  int training_epochs = 100;

  int train_N = 10;
  int test_N = 2;
  int n_visible = 20;
  int n_hidden = 5;

  // training data
  int train_X[10][20] = {
    {1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0},
    {1, 0, 1, 1, 1, 1, 1, 1, 1, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0},
    {1, 1, 0, 1, 1, 1, 1, 1, 0, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0},
    {1, 1, 1, 1, 0, 0, 1, 1, 1, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0},
    {0, 1, 1, 1, 1, 1, 1, 1, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0},
    {0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1},
    {0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 0, 1, 1, 1, 1, 1, 1, 1, 1},
    {0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 1, 0, 1, 1, 1, 1, 1, 0, 1},
    {0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 1, 1, 1, 0, 0, 1, 1, 1, 1},
    {0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 1, 1, 1, 1, 1, 1, 1, 0}
  };

  // construct dA
  dA da_gold, da_h;
  dA__construct(&da_gold, train_N, n_visible, n_hidden, NULL, NULL, NULL);
  dA__construct(&da_h, train_N, n_visible, n_hidden, NULL, NULL, NULL);

  // train using gold standard
  for(epoch=0; epoch<training_epochs; epoch++) {
    for(i=0; i<train_N; i++) {
      dA_train_gold(&da_gold, train_X[i], learning_rate, corruption_level);
    }
  }
  
  // train using kernel
  printf("\nBefore: %f, %f", da_h.W_flat[0], da_h.hbias[0]);
  dA_train_on_device(&da_h, train_X, learning_rate, corruption_level);
  printf("\nAfter: %f, %f\n", da_h.W_flat[0], da_h.hbias[0]);

  // test data
  int test_X[2][20] = {
    {1, 0, 1, 0, 1, 0, 1, 0, 1, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0},
    {0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 1, 0, 1, 1, 0, 1, 1, 0}
  };
  double reconstructed_X[2][20];


  // test
  for(i=0; i<test_N; i++) {
    dA_reconstruct(&da_gold, test_X[i], reconstructed_X[i]);
    for(j=0; j<n_visible; j++) {
      printf("%.5f ", reconstructed_X[i][j]);
    }
    printf("\n");
  }


  // destruct dA
  dA__destruct(&da_gold);
  dA__destruct(&da_h);
}


int main(int argc, char** argv) {
  test_dbn();
  return 0;
}
