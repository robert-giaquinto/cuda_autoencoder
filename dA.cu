#include "hip/hip_runtime.h"
// includes, system
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

// includes, project
#include <cutil.h>

// includes, kernel
#include <dA_kernel.cu>

// declarations
extern "C"
void dA_train_gold(dA*, int*, double, double);




double uniform(double min, double max) {
  return rand() / (RAND_MAX + 1.0) * (max - min) + min;
}


int binomial(int n, double p) {
  if(p < 0 || p > 1) return 0;

  int i;
  int c = 0;
  double r;

  for(i=0; i<n; i++) {
    r = rand() / (RAND_MAX + 1.0);
    if (r < p) c++;
  }

  return c;
}


double sigmoid(double x) {
  return 1.0 / (1.0 + exp(-x));
}


void dA__construct(dA* model, int N, int n_visible, int n_hidden, double **W, double *hbias, double *vbias) {
  int i, j;
  double a = 1.0 / n_visible;

  model->N = N;
  model->n_visible = n_visible;
  model->n_hidden = n_hidden;

  if(W == NULL) {
    model->W = (double **)malloc(sizeof(double*) * n_hidden);
    model->W[0] = (double *)malloc(sizeof(double) * n_visible * n_hidden);
    for(i=0; i<n_hidden; i++) model->W[i] = model->W[0] + i * n_visible;

    for(i=0; i<n_hidden; i++) {
      for(j=0; j<n_visible; j++) {
        model->W[i][j] = uniform(-a, a);
      }
    }
  } else {
    model->W = W;
  }

  if(hbias == NULL) {
    model->hbias = (double *)malloc(sizeof(double) * n_hidden);
    for(i=0; i<n_hidden; i++) model->hbias[i] = 0;
  } else {
    model->hbias = hbias;
  }

  if(vbias == NULL) {
    model->vbias = (double *)malloc(sizeof(double) * n_visible);
    for(i=0; i<n_visible; i++) model->vbias[i] = 0;
  } else {
    model->vbias = vbias;
  }
}

void dA__destruct(dA* model) {
  free(model->W[0]);
  free(model->W);
  free(model->hbias);
  free(model->vbias);
}

void dA_get_corrupted_input(dA* model, int *x, int *tilde_x, double p) {
  int i;
  for(i=0; i<model->n_visible; i++) {
    if(x[i] == 0) {
      tilde_x[i] = 0;
    } else {
      tilde_x[i] = binomial(1, p);
    }
  }
}

// Encode
void dA_get_hidden_values(dA* model, int *x, double *y) {
  int i,j;
  for(i=0; i<model->n_hidden; i++) {
    y[i] = 0;
    for(j=0; j<model->n_visible; j++) {
      y[i] += model->W[i][j] * x[j];
    }
    y[i] += model->hbias[i];
    y[i] = sigmoid(y[i]);
  }
}

// Decode
void dA_get_reconstructed_input(dA* model, double *y, double *z) {
  int i, j;
  for(i=0; i<model->n_visible; i++) {
    z[i] = 0;
    for(j=0; j<model->n_hidden; j++) {
      z[i] += model->W[j][i] * y[j];
    }
    z[i] += model->vbias[i];
    z[i] = sigmoid(z[i]);
  }
}


void dA_reconstruct(dA* model, int *x, double *z) {
  double *y = (double *)malloc(sizeof(double) * model->n_hidden);

  dA_get_hidden_values(model, x, y);
  dA_get_reconstructed_input(model, y, z);

  free(y);
}



void dA_train_on_device(dA*, int*, double, double) {
  // call kernel function from here
}



void test_dbn(void) {
  srand(0);
  int i, j, epoch;

  double learning_rate = 0.1;
  double corruption_level = 0.3;
  int training_epochs = 100;

  int train_N = 10;
  int test_N = 2;
  int n_visible = 20;
  int n_hidden = 5;

  // training data
  int train_X[10][20] = {
    {1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0},
    {1, 0, 1, 1, 1, 1, 1, 1, 1, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0},
    {1, 1, 0, 1, 1, 1, 1, 1, 0, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0},
    {1, 1, 1, 1, 0, 0, 1, 1, 1, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0},
    {0, 1, 1, 1, 1, 1, 1, 1, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0},
    {0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1},
    {0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 0, 1, 1, 1, 1, 1, 1, 1, 1},
    {0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 1, 0, 1, 1, 1, 1, 1, 0, 1},
    {0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 1, 1, 1, 0, 0, 1, 1, 1, 1},
    {0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 1, 1, 1, 1, 1, 1, 1, 0}
  };

  // construct dA
  dA da;
  dA__construct(&da, train_N, n_visible, n_hidden, NULL, NULL, NULL);

  // train using gold standard
  for(epoch=0; epoch<training_epochs; epoch++) {
    for(i=0; i<train_N; i++) {
      dA_train_gold(&da, train_X[i], learning_rate, corruption_level);
    }
  }

  // test data
  int test_X[2][20] = {
    {1, 0, 1, 0, 1, 0, 1, 0, 1, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0},
    {0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 1, 0, 1, 1, 0, 1, 1, 0}
  };
  double reconstructed_X[2][20];


  // test
  for(i=0; i<test_N; i++) {
    dA_reconstruct(&da, test_X[i], reconstructed_X[i]);
    for(j=0; j<n_visible; j++) {
      printf("%.5f ", reconstructed_X[i][j]);
    }
    printf("\n");
  }


  // destruct dA
  dA__destruct(&da);
}


int main(int argc, char** argv) {
  test_dbn();
  return 0;
}
