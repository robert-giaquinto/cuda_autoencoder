#include "hip/hip_runtime.h"
// includes, system
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

// includes, project
#include <cutil.h>

// includes, kernel
#include <dA_kernel1.cu>
//#include <dA.h>


// declarations for CPU train functions
extern "C"
void dA_train_gold(dA*, int*, double, double);
void dA_get_hidden_values(dA*, int*, double*);
void dA_get_reconstructed_input(dA*, double*, double*);
//**
void dA_get_corrupted_input(dA*, int*, int*, double);
void dA_get_hidden_values(dA*, int*, double*);
void dA_get_reconstructed_input(dA*, double*, double*);
int binomial(int n, double p);
double sigmoid(double x);
//**

// functions defined in this file are for intializing the autoencoder
double uniform(double min, double max);
void dA__construct(dA *model, int N, int n_visible, int n_hidden, double **W, double *hbias, double *vbais);
void dA__destruct(dA *model);
void dA_reconstruct(dA *model, int *x, double *z);
void test_dbn();
void dA_train_on_device(dA*, int[][N_FEATS], double, double);

 // Temporary testing

//* end of temporary testing
// Begin definign functions
double uniform(double min, double max) {
  return rand() / (RAND_MAX + 1.0) * (max - min) + min;
}


void dA__construct(dA* model, int N, int n_visible, int n_hidden, double **W, double *hbias, double *vbias) {
  int i, j;
  double a = 1.0 / n_visible;

  model->N = N;
  model->n_visible = n_visible;
  model->n_hidden = n_hidden;

  if(W == NULL) {
    model->W = (double **)malloc(sizeof(double*) * n_hidden);
    model->W_flat = (double*)malloc(sizeof(double)*n_hidden*n_visible);
    model->W[0] = (double *)malloc(sizeof(double) * n_visible * n_hidden);
    for(i=0; i<n_hidden; i++) model->W[i] = model->W[0] + i * n_visible;

    for(i=0; i<n_hidden; i++) {
      for(j=0; j<n_visible; j++) {
        double u = uniform(-a, a);
        model->W_flat[i*n_visible + j] = u;
        model->W[i][j] = u;
      }
    }
  } else {
    model->W = W;
  }

  if(hbias == NULL) {
    model->hbias = (double *)malloc(sizeof(double) * n_hidden);
    for(i=0; i<n_hidden; i++) model->hbias[i] = 0;
  } else {
    model->hbias = hbias;
  }

  if(vbias == NULL) {
    model->vbias = (double *)malloc(sizeof(double) * n_visible);
    for(i=0; i<n_visible; i++) model->vbias[i] = 0;
  } else {
    model->vbias = vbias;
  }
}


void dA__destruct(dA* model) {
  //free(model->W[0]);
  free(model->W);
  free(model->W_flat);
  free(model->hbias);
  free(model->vbias);
}


void dA_reconstruct(dA* model, int *x, double *z) {
  double *y = (double *)malloc(sizeof(double) * model->n_hidden);

  dA_get_hidden_values(model, x, y);
  dA_get_reconstructed_input(model, y, z);

  free(y);
}


int* flatten_array(int arr[N_OBS][N_FEATS]) {
  int *flat = (int *)malloc(sizeof(int) * N_OBS * N_FEATS);
  for (int i=0; i < N_OBS; ++i) {
    for (int j=0; j < N_FEATS; ++j) {
      flat[i*N_FEATS + j] = arr[i][j];
    }
  }
  return flat;
}

double* flatten_w(double **W, int n_visible, int n_hidden) {
  double *flat = (double *)malloc(sizeof(double) * n_visible * n_hidden);
  for (int i=0; i < n_hidden; ++i) {
    for (int j=0; j < n_visible; ++j) {
      flat[i*n_visible + j] = W[i][j];
    }
  }
  return flat;
}

int * allocate_device_x() {
  int *x_d = NULL;
  int size = N_OBS * N_FEATS * sizeof(int);
  hipMalloc((void**)&x_d, size);
  return x_d;
}


int * allocate_device_tile_x(int m) {
  int *tile_x_d = NULL;
  int size = m * sizeof(int);
  hipMalloc((void**)&tile_x_d, size);
  return tile_x_d;
}


double * allocate_device_z(int m) {
  double *z_d = NULL;
  int size = m * sizeof(double);
  hipMalloc((void**)&z_d, size);
  return z_d;
}

double * allocate_device_y(int n) {
  double *y_d = NULL;
  int size = n * sizeof(double);
  hipMalloc((void**)&y_d, size);
  return y_d;
}

double*  allocate_device_dW() {
  double *dW_flat;
  int dW_size = sizeof(double) * N_HIDDEN * N_FEATS;
  hipMalloc((void**)&dW_flat, dW_size);
  return dW_flat;
  
}

double* allocate_device_dhbias() {
  double *dhbias;
  int dhbias_size = sizeof(double) * N_HIDDEN;
  hipMalloc((void**)&dhbias, dhbias_size);
  return dhbias;
}

double* allocate_device_dvbias() {
  double *dvbias;
  int dvbias_size = sizeof(double) * N_FEATS;
  hipMalloc((void**)&dvbias, dvbias_size);
  return dvbias;  
}

double* allocate_device_dL_vbias(int m, int n) {
  double *dL_vbias;
  int dL_vbias_size = sizeof(double) * m * n;
  hipMalloc((void**)&dL_vbias, dL_vbias_size);
  return dL_vbias;  
}

double* allocate_device_dL_hbias(int m,int n) {
  double *dL_hbias;
  int dL_hbias_size = sizeof(double) * m * n;
  hipMalloc((void**)&dL_hbias, dL_hbias_size);
  return dL_hbias;  
}

void copy_x_to_device(int *x_d, int *x_h) {
  int size = N_OBS * N_FEATS * sizeof(int);
  hipMemcpy(x_d, x_h, size, hipMemcpyHostToDevice);
}

void copy_x_to_host(int *x_d, int *x_h) {
  int size = N_OBS * N_FEATS * sizeof(int);
  hipMemcpy(x_h, x_d, size, hipMemcpyDeviceToHost);
}

void dA_get_corrupted_input_miniBatch(int n_visible, int *x, int *tilde_x, double p,int ib,int batchsize) {
  for (int j=0;j<batchsize;j++){
  	for(int i=0; i<n_visible; i++) {
   	 if(x[ib*batchsize+j*n_visible+i] == 0) {
   	   tilde_x[i*batchsize+j] = 0;
   	 } else {
  	    tilde_x[i*batchsize+j] = binomial(1, p);
   	 }
  	}
  }
}

void dA_train_on_device1(dA *model_h, int train_X[N_OBS][N_FEATS], double lr, double corruption_level,int training_epochs) {
  //
  hipError_t cuda_ret;
  int epoch;
  int offsetXval;
  //int i,j;
  printf("\n ** in device processing ** \n"); 
  //
  float time1, time2, time3,time31,time32,time33,time34,time35,time36,time37,time38,time39,time322;
  time1=time2=time3=time31=time32=time33=time34=time35=time36=time37=time38=time39=time322=0.0;
  unsigned int timer1; cutCreateTimer(&timer1); cutStartTimer(timer1);  
  //
  double p = 1 - corruption_level;  
  // flatten input array
  int *X_h = flatten_array(train_X);
  // allocate space on device
  int *X_d = allocate_device_x();
  int *tilde_x_d = allocate_device_tile_x(BATCHSIZE*N_FEATS);
  int *tilde_x_h = (int*)malloc(sizeof(int)* BATCHSIZE * N_FEATS);
  //double *y_h = (double*)malloc(sizeof(double)* N_HIDDEN);
  double *yb_h = (double*)malloc(sizeof(double)* N_HIDDEN*BATCHSIZE);
  double *yb_d = allocate_device_y(N_HIDDEN*BATCHSIZE); 
  //double *y_d = allocate_device_y(N_HIDDEN); 
  double *hW_flat = (double*)malloc(sizeof(double)* N_HIDDEN * N_FEATS);
  double *dW_flat = allocate_device_dW();
  double *dhbias = allocate_device_dhbias();
  double *dvbias = allocate_device_dvbias(); 
  double *z_d = allocate_device_z(N_FEATS*BATCHSIZE);
  double *z_h = (double*)malloc(sizeof(double)*BATCHSIZE* N_FEATS);
   //
  double *L_vbias = (double *)malloc(sizeof(double) * BATCHSIZE * N_FEATS);
  double *dL_vbias = allocate_device_dL_vbias(BATCHSIZE,N_FEATS);
  double *L_hbias = (double *)malloc(sizeof(double) * BATCHSIZE * N_HIDDEN);
  double *dL_hbias = allocate_device_dL_hbias(BATCHSIZE, N_HIDDEN);
  //
  // initialize a random state for each thread;
  //hiprandState *d_state;
  //hipMalloc(&d_state, N_FEATS * BATCHSIZE);
  //
  cutStopTimer(timer1); time1 = cutGetTimerValue(timer1); cutDeleteTimer(timer1);
  //
  unsigned int timer2;cutCreateTimer(&timer2); cutStartTimer(timer2);
  // copy data over to device
  copy_x_to_device(X_d, X_h);
  //copy_x_to_device(tilde_x_d, X_h);
  //dA model_d = init_device_ae(*model_h);
  // copy over data
  hipMemcpy(dW_flat, model_h->W_flat, sizeof(double)*N_HIDDEN*N_FEATS, hipMemcpyHostToDevice);
  hipMemcpy(dhbias, model_h->hbias, sizeof(double)*N_HIDDEN, hipMemcpyHostToDevice);
  hipMemcpy(dvbias, model_h->vbias, sizeof(double)*N_FEATS, hipMemcpyHostToDevice);
  //
  cutStopTimer(timer2); time2 = cutGetTimerValue(timer2); cutDeleteTimer(timer2);
  //
  //printf("X_h %d %d",X_h[1],X_h[2]);
  //
  //unsigned int timer3; cutCreateTimer(&timer3); cutStartTimer(timer3);
  //
  for(epoch=0; epoch<training_epochs; epoch++) {

  	int n_batches = ceil(N_OBS / BATCHSIZE); 

  	//2. encode to get hidden values y
  	dim3 dimGrid2(1);
  	dim3 dimBlock2(BATCHSIZE);
  	//dim3 dimBlock2(1);
  	//printf("\n N : %d",model_h->N);
  	int ib=0;
  	//n_batches = 1;
  	for (ib=0; ib<n_batches;ib++) {
		//  copy_ae_to_device(model_d, model_h);
  		unsigned int timer31; cutCreateTimer(&timer31);	cutStartTimer(timer31);
  		//1. set up corrupted input for all together
  		//printf("Batches %d %d",BATCHSIZE,n_batches);
		//int n31Threads  = 400; // 20 * n_feats say
		//int n31Blocks = ceil (model_h->n_visible*BATCHSIZE / n31Threads);
		dim3 dimGrid31(BATCHSIZE);
     		dim3 dimBlock31(N_FEATS);
		offsetXval = ib*BATCHSIZE*N_FEATS;
     		//dA_get_corrupted_input_kernel<<<dimGrid31, dimBlock31>>>(BATCHSIZE*N_FEATS, X_d, tilde_x_d, p, offsetXval,d_state);
     		dA_get_corrupted_input_kernel<<<dimGrid31, dimBlock31>>>(BATCHSIZE*N_FEATS, X_d, tilde_x_d, p, offsetXval);
  		cuda_ret = hipDeviceSynchronize();
 		if (cuda_ret != hipSuccess) 	printf("Error in kernel");
		
		//dA_get_corrupted_input(model_h, X_h, tilde_x_h, p);
  		//hipMemcpy(tilde_x_h, tilde_x_d, sizeof(int) * BATCHSIZE * N_FEATS, hipMemcpyDeviceToHost);
		//dA_get_corrupted_input_miniBatch(model_h->n_visible, X_h, tilde_x_h, p,ib,BATCHSIZE);
  		//hipMemcpy(tilde_x_d, tilde_x_h, sizeof(int) * BATCHSIZE * model_h->n_visible, hipMemcpyHostToDevice);
  		// printf("\ntilde_x_h : "); for(int j=0;j<BATCHSIZE*model_h->n_visible;j++){ printf(" %d ",tilde_x_h[j]); }
		
  		//copy_x_to_host(tilde_x, X_h);
  		cutStopTimer(timer31); time31 += cutGetTimerValue(timer31); cutDeleteTimer(timer31);
  		//hipDeviceSynchronize();
		//
        	//2. encode to get hidden values y
  		unsigned int timer32; cutCreateTimer(&timer32);	cutStartTimer(timer32);
		//
		int n32Threads = TILE_WIDTH;
		int n32Blocks = N_HIDDEN / n32Threads;
		int m32Blocks = BATCHSIZE / n32Threads;
		if (N_HIDDEN % n32Threads) n32Blocks++;
		if (BATCHSIZE % n32Threads) m32Blocks++;
		dim3 dimGrid32(m32Blocks, n32Blocks);
 		//dim3 dimGrid32(n32Blocks, m32Blocks);
		dim3 dimBlock32(n32Threads,n32Threads);
       		//dA_get_hidden_values_kernel<<<dimGrid32,dimBlock32>>>(N_HIDDEN,N_FEATS,dW_flat,dhbias,tilde_x_d,y_d,ib);
       		//dA_get_hidden_values_kernel1<<<dimGrid32,dimBlock32>>>(N_HIDDEN,N_FEATS,dW_flat,dhbias,tilde_x_d,y_d,ib);
       		dA_get_hidden_values_kernel<<<dimGrid32,dimBlock32>>>(N_HIDDEN,N_FEATS,dW_flat,dhbias,tilde_x_d,yb_d,ib);
  		cuda_ret = hipDeviceSynchronize();
 		if (cuda_ret != hipSuccess)
      			printf("Error in kernel");

  		cutStopTimer(timer32); time32 += cutGetTimerValue(timer32); cutDeleteTimer(timer32);
  		//hipMemcpy(yb_h, yb_d,sizeof(double) * N_HIDDEN*BATCHSIZE, hipMemcpyDeviceToHost);
		//printf("\nyb_h : "); for(int j=0;j<N_HIDDEN*BATCHSIZE;j++){	printf(" %f ",yb_h[j]); } 
    		//3.decode by reconstrution to get z
  		unsigned int timer33; cutCreateTimer(&timer33);	cutStartTimer(timer33);
		//
		int n33Threads = TILE_WIDTH;
		int n33Blocks = N_FEATS / n33Threads;
		int m33Blocks = BATCHSIZE / n33Threads;
		if (N_FEATS % n32Threads) n33Blocks++;
		if (BATCHSIZE % n33Threads) m33Blocks++;
		dim3 dimGrid33(m33Blocks, n33Blocks);
		dim3 dimBlock33(n33Threads,n33Threads);
    		 //dA_get_reconstructed_input_kernel<<<dimGrid2,dimBlock2>>>(N_HIDDEN,N_FEATS,dW_flat,dvbias,z_d,y_d,ib,BATCHSIZE);
    		dA_get_reconstructed_input_kernel<<<dimGrid33,dimBlock33>>>(N_HIDDEN,N_FEATS,dW_flat,dvbias,z_d,yb_d,ib,BATCHSIZE);
  		cuda_ret = hipDeviceSynchronize();
 		if (cuda_ret != hipSuccess)
      			printf("Error in kernel");
  		//hipMemcpy(z_h, z_d,sizeof(double) * N_FEATS * BATCHSIZE, hipMemcpyDeviceToHost);
  		//printf("\nz_h: "); for(int j=0;j<N_FEATS*BATCHSIZE;j++){ printf(" %f ",z_h[j]); }
  		cutStopTimer(timer33); time33 += cutGetTimerValue(timer33); cutDeleteTimer(timer33);
    		//4. Update error in reconstruction - visible error for every minibatch by atomic add kernel
  		unsigned int timer34; cutCreateTimer(&timer34);	cutStartTimer(timer34);
		dim3 dimGrid34(BATCHSIZE);
     		dim3 dimBlock34(N_FEATS);
    		dA_L_vbias_kernel<<<dimGrid34,dimBlock34>>>(model_h->N,dL_vbias,dvbias,N_FEATS,X_d,z_d,offsetXval,BATCHSIZE,lr);
  		cuda_ret = hipDeviceSynchronize();
 		if (cuda_ret != hipSuccess)
      			printf("Error in kernel");
		//hipMemcpy(L_vbias, dL_vbias,sizeof(double) *N_FEATS*BATCHSIZE, hipMemcpyDeviceToHost);
		//printf("\nL_vbias: "); for(int j=0;j<N_FEATS*BATCHSIZE;j++){ printf(" %f ",L_vbias[j]); }				
  		cutStopTimer(timer34); time34 += cutGetTimerValue(timer34); cutDeleteTimer(timer34);
      		//5.Update error in hidden units outputs, we would use it to update weights
  		unsigned int timer35; cutCreateTimer(&timer35);	cutStartTimer(timer35);
		int n35Threads = TILE_WIDTH;
		int n35Blocks = N_HIDDEN / n35Threads;
		int m35Blocks = BATCHSIZE / n35Threads;
		if (N_HIDDEN % n35Threads) n35Blocks++;
		if (BATCHSIZE % n35Threads) m35Blocks++;
		dim3 dimGrid35(m32Blocks, n32Blocks);
		dim3 dimBlock35(n35Threads,n35Threads);
    		dA_L_hbias_kernel<<<dimGrid35,dimBlock35>>>(model_h->N,dL_vbias,dL_hbias,dhbias,N_HIDDEN,N_FEATS,yb_d,dW_flat,ib,BATCHSIZE,lr);
  		cuda_ret = hipDeviceSynchronize();
 		if (cuda_ret != hipSuccess)
      			printf("Error in kernel");
		//hipMemcpy(L_hbias, dL_hbias,sizeof(double) *N_HIDDEN*BATCHSIZE, hipMemcpyDeviceToHost);
		//printf("\nL_hbias: "); for(int j=0;j<N_HIDDEN*BATCHSIZE;j++){ printf(" %f ",L_hbias[j]); }				
  		cutStopTimer(timer35); time35 += cutGetTimerValue(timer35); cutDeleteTimer(timer35);
    		//6. Weights updates for minibatch
  		unsigned int timer36; cutCreateTimer(&timer36);	cutStartTimer(timer36);
  		     //2. encode to get hidden values y
  		dim3 dimGrid36(N_HIDDEN);
  		dim3 dimBlock36(N_FEATS);
    		dA_W_kernel<<<dimGrid36,dimBlock36>>>(model_h->N,dL_vbias,dL_hbias,model_h->n_hidden,model_h->n_visible,
							yb_d,dW_flat,tilde_x_d,ib,BATCHSIZE,lr);
  		cuda_ret = hipDeviceSynchronize();
 		if (cuda_ret != hipSuccess)
      			printf("Error in kernel");
  		//hipMemcpy(hW_flat, dW_flat,sizeof(double) * N_HIDDEN * N_FEATS, hipMemcpyDeviceToHost);
		//printf("\nhW_flat: "); for(int j=0;j<N_HIDDEN*N_FEATS;j++){ printf(" %f ",hW_flat[j]); }				
  		cutStopTimer(timer36); time36 += cutGetTimerValue(timer36); cutDeleteTimer(timer36);
 	}
	//******************************************************************************************************

  }
  //
  //
  unsigned int timer37; cutCreateTimer(&timer37);cutStartTimer(timer37);
  cuda_ret = hipDeviceSynchronize();
  if (cuda_ret != hipSuccess)
      printf("Error in kernel");
  cutStopTimer(timer37); time37 += cutGetTimerValue(timer37); cutDeleteTimer(timer37);
  //
  unsigned int timer38; cutCreateTimer(&timer38);cutStartTimer(timer38);
  //hipMemcpy(tilde_x_h, tilde_x_d,sizeof(double) * N_OBS * N_FEATS, hipMemcpyDeviceToHost);
  //hipMemcpy(yb_h, yb_d,sizeof(double) * N_HIDDEN*BATCHSIZE, hipMemcpyDeviceToHost);
  //printf("\nyb_h : "); for(int j=0;j<N_HIDDEN*BATCHSIZE;j++){ printf(" %f ",yb_h[j]); }

  //hipMemcpy(z_h, z_d,sizeof(double) * 1*N_FEATS, hipMemcpyDeviceToHost);
  //hipMemcpy(L_vbias, dL_vbias,sizeof(double) * 1*N_FEATS, hipMemcpyDeviceToHost);
  hipMemcpy(model_h->vbias, dvbias,sizeof(double) * N_FEATS, hipMemcpyDeviceToHost);
  //hipMemcpy(L_hbias, dL_hbias,sizeof(double) * 1*N_HIDDEN, hipMemcpyDeviceToHost);
  hipMemcpy(model_h->hbias, dhbias,sizeof(double) * N_HIDDEN, hipMemcpyDeviceToHost);
  hipMemcpy(model_h->W_flat, dW_flat,sizeof(double) * N_HIDDEN * N_FEATS, hipMemcpyDeviceToHost);
  //
  //*
  //printf("ibb is: %d\n",ib);
  //for(int i=0;i<N_OBS;i++) {
  //    printf("\ntile_x_h : "); for(int j=0;j<5;j++){ printf(" %f ",tilde_x_h[i*N_OBS+j]); }
  //}

  //for(int i=0;i<N_OBS;i++) {
  //printf("\nz_h: "); for(int j=0;j<5;j++){ printf(" %f ",z_h[j]); }
  //}
  //printf("\nh vbias: "); for(int j=0;j<N_FEATS;j++){ printf(" %f ",model_h->vbias[j]); }
  //printf("\nh hbias: "); for(int j=0;j<N_HIDDEN;j++){ printf(" %f ",model_h->hbias[j]); }
  //printf("\nh Weights: ");for(int j=0;j<N_HIDDEN*N_FEATS;j++){ printf(" %f ",model_h->W_flat[j]); }
  //*/
  //
  cutStopTimer(timer38); time38 += cutGetTimerValue(timer38); cutDeleteTimer(timer38);
  //hipMemcpy(model_h->W, dW_flat,sizeof(double) * N_HIDDEN * N_FEATS, hipMemcpyDeviceToHost);
  unsigned int timer39; cutCreateTimer(&timer39);	cutStartTimer(timer39);
  //We can not directly copy to W and W is used to test, so we populate it using a loop
  for(int i=0; i<model_h->n_hidden; i++) {
      for(int j=0; j<model_h->n_visible; j++) {
       	model_h->W[i][j] = model_h->W_flat[i*model_h->n_visible + j];
      }	
  }
  //
  cutStopTimer(timer39); time39 += cutGetTimerValue(timer39); cutDeleteTimer(timer39);

  //cutStopTimer(timer3);time3 = cutGetTimerValue(timer3); cutDeleteTimer(timer3);
  //
  // free up memory
  hipFree(X_d); hipFree(tilde_x_d); hipFree(dW_flat);
  hipFree(dhbias); hipFree(dvbias); hipFree(dL_vbias);
  hipFree(dL_hbias); //hipFree(y_d); 
  hipFree(z_d); 
  //free(y_h);
  X_d = NULL;tilde_x_d = NULL; dW_flat = NULL; dhbias = NULL;
  dvbias = NULL; dL_vbias = NULL;dL_hbias = NULL;
  //y_d = NULL; 
  z_d = NULL; 

  //
  free(L_hbias);free(L_vbias);free(z_h);
  free(yb_h); yb_h = NULL;
  //free(y_h); y_h = NULL;
  free(tilde_x_h); tilde_x_h = NULL;
  free(hW_flat); hW_flat = NULL;
   //
  printf("\ntime1  : %f\n", time1);
  printf("time2  : %f\n", time2);
  printf("time3  : %f\n", time3);
  printf("time31 : %f\n", time31);
  printf("time32 : %f\n", time32);
  printf("time322 : %f\n", time322);
  printf("time33 : %f\n", time33);
  printf("time34 : %f\n", time34);
  printf("time35 : %f\n", time35);
  printf("time36 : %f\n", time36);
  printf("time37 : %f\n", time37);
  printf("time38 : %f\n", time38);
  printf("time39 : %f\n", time39);
  //
  
}


void test_dbn(void) {
  srand(0);
  int i, j, epoch;
  float device_time;
  float host_time;

  double learning_rate = 0.1;
  double corruption_level = 0.3;
  //int training_epochs = 100;
  int training_epochs = 100;

  //int train_N = 10;
  int train_N = N_OBS;
  int test_N = 2;
  int n_visible = N_FEATS;
  int n_hidden = N_HIDDEN;

  // training data
  /*
  int train_X[10][20] = {
    {1, 1, 1, 1, 1, 1, 1, 1, 1, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0},
    {1, 0, 1, 1, 1, 1, 1, 1, 1, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0},
    {1, 1, 0, 1, 1, 1, 1, 1, 0, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0},
    {1, 1, 1, 1, 0, 0, 1, 1, 1, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0},
    {0, 1, 1, 1, 1, 1, 1, 1, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0},
    {0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 1, 1, 1, 1, 1, 1, 1, 1, 1},
    {0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 0, 1, 1, 1, 1, 1, 1, 1, 1},
    {0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 1, 0, 1, 1, 1, 1, 1, 0, 1},
    {0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 1, 1, 1, 0, 0, 1, 1, 1, 1},
    {0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 1, 1, 1, 1, 1, 1, 1, 0}
  };
  */
  //*
  //  To increase size of array for testing
  int train_X[N_OBS][N_FEATS];
  for (int i1=0;i1<N_OBS;i1++) {
   for (int i2=0;i2<N_FEATS;i2++) {
        train_X[i1][i2] = rand() % 2;
     }
   }
  printf("training input : %f %f\n",train_X[0][0],train_X[N_OBS-1][N_FEATS-1]);
  //*/
  //

  // construct dA
  dA da_gold, da_h;
  dA__construct(&da_gold, train_N, n_visible, n_hidden, NULL, NULL, NULL);
  dA__construct(&da_h, train_N, n_visible, n_hidden, NULL, NULL, NULL);
  //memcpy(da_h.W, da_gold.W, sizeof(double)*n_hidden*n_visible);
  //memcpy(da_h.W_flat, da_gold.W_flat,sizeof(double)*n_hidden*n_visible);
  for (int i=0;i<n_hidden;i++) {
    for (int j=0;j<n_visible;j++) {
	da_h.W[i][j] = da_gold.W[i][j];
	da_h.W_flat[i*n_visible+j] = da_gold.W_flat[i*n_visible+j];
    }
  }
  //
  //*** to compare, initial values should be same for both the objects
  printf("da_gold W      : %f %f %f \n", da_gold.W[0][0],da_gold.W[0][1],da_gold.W[0][2]);
  printf("da_h W         : %f %f %f \n", da_h.W[0][0],da_h.W[0][1],da_h.W[0][2]);
  printf("da_gold W_flat : %f %f %f \n", da_gold.W_flat[0],da_gold.W_flat[1],da_gold.W_flat[2]);
  printf("da_h W_flat    : %f %f %f \n", da_h.W_flat[0],da_h.W_flat[1],da_h.W_flat[2]);
  printf("da_gold n_visi : %d \n", da_gold.n_visible);
  printf("da_h n_visible : %d \n", da_h.n_visible);
  printf("da_gold hbias  : %f %f %f \n",da_gold.hbias[0],da_gold.hbias[1],da_gold.hbias[2]);
  printf("da_h hbias     : %f %f %f \n",da_h.hbias[0],da_h.hbias[1],da_h.hbias[2]);
  printf("da_gold vbias  : %f %f %f \n",da_gold.vbias[0],da_gold.vbias[1],da_gold.vbias[2]);
  printf("da_h vbias     : %f %f %f \n",da_h.vbias[0],da_h.vbias[1],da_h.vbias[2]);
  //***
  printf("  Starting gold training..");
  unsigned int cputimer;
  cutCreateTimer(&cputimer);
  cutStartTimer(cputimer);
  // train using gold standard
  for(epoch=0; epoch<training_epochs; epoch++) {
    for(i=0; i<train_N; i++) {
      dA_train_gold(&da_gold, train_X[i], learning_rate, corruption_level);
    }
  }
  //
  cutStopTimer(cputimer);
  host_time = cutGetTimerValue(cputimer);
  cutDeleteTimer(cputimer);
  printf("Ending gold training..");
  //

  //* Start of GPU Kernel Call Code
  printf("Starting device training..");
  unsigned int gputimer;
  cutCreateTimer(&gputimer);
  cutStartTimer(gputimer);
  // train using kernel
  //dA_train_on_device(&da_h, train_X, learning_rate, corruption_level);
  dA_train_on_device1(&da_h, train_X, learning_rate, corruption_level,training_epochs);
  //
  cutStopTimer(gputimer);
  device_time = cutGetTimerValue(gputimer);
  cutDeleteTimer(gputimer);
  printf("Ending device training..");
  /*
  //
  printf("\nCPU Weights:"); for(int j=0;j<5;j++) {printf("%f ", da_gold.W_flat[j]);};
  printf("\nGPU Weights:"); for(int j=0;j<5;j++) {printf("%f ", da_h.W_flat[j]);};
  //
  //
  printf("\nCPU hbias:"); for(int j=0;j<5;j++) {printf("%f ", da_gold.hbias[j]);};
  printf("\nGPU hbias:"); for(int j=0;j<5;j++) {printf("%f ", da_h.hbias[j]);};
  //
  //
  printf("\nCPU vbias:"); for(int j=0;j<5;j++) {printf("%f ", da_gold.vbias[j]);};
  printf("\nGPU vbias:"); for(int j=0;j<5;j++) {printf("%f ", da_h.vbias[j]);};
  //
  */
  //* End of GPU Coode
  
  // test data
  /*
  int test_X[2][20] = {
    {1, 0, 1, 0, 1, 0, 1, 0, 1, 1, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0},
    {0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1, 1, 0, 1, 1, 0, 1, 1, 0}
  };
  */
  //*  To increase size of array for testing
  int test_X[2][N_FEATS];
  for (int i1=0;i1<test_N;i1++) {
   for (int i2=0;i2<N_FEATS;i2++) {
        train_X[i1][i2] = rand() % 2;
     }
   }
  //*/
  double reconstructed_X[2][N_FEATS];

  printf("\n : CPU test now: \n");
  // test CPU using &da_gold object
  for(i=0; i<test_N; i++) {
    dA_reconstruct(&da_gold, test_X[i], reconstructed_X[i]);
    for(j=0; j<n_visible; j++) { printf("%.5f ", reconstructed_X[i][j]);}
    printf("\n");
  }

   printf("\n : GPU test now: \n");
  // test GPU using &da_h object
  for(i=0; i<test_N; i++) {
    dA_reconstruct(&da_h, test_X[i], reconstructed_X[i]);
    for(j=0; j<n_visible; j++) {printf("%.5f ", reconstructed_X[i][j]);}
    printf("\n");
  }

  // destruct dA
  dA__destruct(&da_gold);
  dA__destruct(&da_h);
    //
  printf("Host time          : %f\n", host_time);
  printf("Device time        : %f\n", device_time);
  printf("Speedup host/device: %fX\n", host_time/device_time);
  printf("***testing over***\n");

}


int main(int argc, char** argv) {
  test_dbn();
  return 0;
}
